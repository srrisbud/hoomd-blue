#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2014 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


// Maintainer: ndtrung

#include "RigidData.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file RigidData.cu

*/


#ifdef NVCC
//! Kernel for seting R and V of rigid body particles
/*!
    \param pdata_pos Particle position
    \param pdata_vel Particle velocity
    \param pdata_image Particle image
    \param pdata_orientation Particle orientation
    \param d_pgroup_idx Particle index
    \param n_pgroup Number of particles in the group
    \param d_particle_offset Local index of a particle in the body
    \param d_particle_body Body index of a particle
    \param d_rigid_orientation Body orientation (quaternion)
    \param d_rigid_com Body center of mass
    \param d_rigid_vel Body velocity
    \param d_rigid_angvel Body angular velocity
    \param d_rigid_image Body image
    \param d_rigid_particle_dis Position of a particle in the body frame
    \param d_rigid_particle_orientation Orientation of a particle in the body frame
    \param nmax Maximum number of particles per body
    \param box Box dimensions for periodic boundary condition handling
*/
template<bool set_x>
__global__ void gpu_rigid_setRV_kernel(Scalar4* pdata_pos,
                                       Scalar4* pdata_vel,
                                       int3* pdata_image,
                                       Scalar4* pdata_orientation,
                                       unsigned int *d_pgroup_idx,
                                       unsigned int n_pgroup,
                                       unsigned int *d_particle_offset,
                                       unsigned int *d_particle_body,
                                       Scalar4* d_rigid_orientation,
                                       Scalar4* d_rigid_com,
                                       Scalar4* d_rigid_vel,
                                       Scalar4* d_rigid_angvel,
                                       int3* d_rigid_image,
                                       Scalar4* d_rigid_particle_dis,
                                       Scalar4* d_rigid_particle_orientation,
                                       unsigned int nmax,
                                       BoxDim box)
    {
    Scalar4 com, vel, angvel, ex_space, ey_space, ez_space;
    int3 body_image = make_int3(0, 0, 0);

    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx >= n_pgroup)
        return;

    unsigned int pidx = d_pgroup_idx[group_idx];
    unsigned int idx_body = d_particle_body[pidx];
    unsigned int particle_offset = d_particle_offset[pidx];
    Scalar4 body_orientation = d_rigid_orientation[idx_body];

    com = d_rigid_com[idx_body];
    vel = d_rigid_vel[idx_body];
    angvel = d_rigid_angvel[idx_body];
    if (set_x)
        {
        body_image = d_rigid_image[idx_body];
        }

    exyzFromQuaternion(body_orientation, ex_space, ey_space, ez_space);

    int localidx = idx_body * nmax + particle_offset;
    Scalar4 particle_pos = d_rigid_particle_dis[localidx];
    Scalar4 constituent_orientation = d_rigid_particle_orientation[localidx];

    // compute ri with new orientation
    Scalar3 ri;
    ri.x = ex_space.x * particle_pos.x + ey_space.x * particle_pos.y + ez_space.x * particle_pos.z;
    ri.y = ex_space.y * particle_pos.x + ey_space.y * particle_pos.y + ez_space.y * particle_pos.z;
    ri.z = ex_space.z * particle_pos.x + ey_space.z * particle_pos.y + ez_space.z * particle_pos.z;

    Scalar3 ppos;
    int3 image;
    Scalar4 porientation;
    if (set_x)
        {
        // x_particle = com + ri
        ppos.x = com.x + ri.x;
        ppos.y = com.y + ri.y;
        ppos.z = com.z + ri.z;

        // time to fix the periodic boundary conditions
        image = body_image;
        box.wrap(ppos, image);

        // update particle orientation
        quatquat(body_orientation,
                 constituent_orientation,
                 porientation);
        }

    // v_particle = vel + angvel x ri
    Scalar4 pvel = pdata_vel[pidx];
    pvel.x = vel.x + angvel.y * ri.z - angvel.z * ri.y;
    pvel.y = vel.y + angvel.z * ri.x - angvel.x * ri.z;
    pvel.z = vel.z + angvel.x * ri.y - angvel.y * ri.x;

    // write out the results
    if (set_x)
        {
        pdata_pos[pidx] = make_scalar4(ppos.x, ppos.y, ppos.z, pdata_pos[pidx].w);
        pdata_image[pidx] = image;
        pdata_orientation[pidx] = porientation;
        }
    pdata_vel[pidx] = pvel;
    }
#endif

// Sets R and v of particles of the rigid body on the GPU
/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_image array of particle images
    \param d_body array of particle body ids
    \param rigid_data Rigid body data
    \param d_pdata_orientation Particle orientations
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate (all particles in rigid bodies)
    \param group_size Number of members in the group
    \param box Box dimensions for periodic boundary condition handling
    \param set_x boolean indicating whether the positions are changed or not (first or second step of integration)
*/
hipError_t gpu_rigid_setRV(Scalar4 *d_pos,
                            Scalar4 *d_vel,
                            int3 *d_image,
                            unsigned int *d_body,
                                   const gpu_rigid_data_arrays& rigid_data,
                                   Scalar4 *d_pdata_orientation,
                                   unsigned int *d_group_members,
                                   unsigned int group_size,
                                   const BoxDim& box,
                                   bool set_x)
    {

    assert(d_pos);
    assert(d_vel);
    assert(d_pdata_orientation);
    assert(d_image);
    assert(d_group_members);

    assert(rigid_data.particle_offset);
    assert(d_body);
    assert(rigid_data.orientation);
    assert(rigid_data.com);
    assert(rigid_data.vel);
    assert(rigid_data.angvel);
    assert(rigid_data.body_image);
    assert(rigid_data.particle_pos);
    assert(rigid_data.particle_orientation);

    unsigned int nmax = rigid_data.nmax;

    unsigned int block_size = 192;
    dim3 particle_grid(group_size/block_size+1, 1, 1);
    dim3 particle_threads(block_size, 1, 1);

    if (set_x)
        gpu_rigid_setRV_kernel<true><<< particle_grid, particle_threads >>>(d_pos,
                                                                        d_vel,
                                                                        d_image,
                                                                        d_pdata_orientation,
                                                                        d_group_members,
                                                                        group_size,
                                                                        rigid_data.particle_offset,
                                                                        d_body,
                                                                        rigid_data.orientation,
                                                                        rigid_data.com,
                                                                        rigid_data.vel,
                                                                        rigid_data.angvel,
                                                                        rigid_data.body_image,
                                                                        rigid_data.particle_pos,
                                                                        rigid_data.particle_orientation,
                                                                        nmax,
                                                                        box);
     else
        gpu_rigid_setRV_kernel<false><<< particle_grid, particle_threads >>>(d_pos,
                                                                        d_vel,
                                                                        d_image,
                                                                        d_pdata_orientation,
                                                                        d_group_members,
                                                                        group_size,
                                                                        rigid_data.particle_offset,
                                                                        d_body,
                                                                        rigid_data.orientation,
                                                                        rigid_data.com,
                                                                        rigid_data.vel,
                                                                        rigid_data.angvel,
                                                                        rigid_data.body_image,
                                                                        rigid_data.particle_pos,
                                                                        rigid_data.particle_orientation,
                                                                        nmax,
                                                                        box);
        return hipSuccess;
}

//! Kernel driven by gpu_compute_virial_correction_end()
__global__ void gpu_compute_virial_correction_end_kernel(Scalar *d_net_virial,
                                                         unsigned int virial_pitch,
                                                         const Scalar4 *d_net_force,
                                                         const Scalar4 *d_oldpos,
                                                         const Scalar4 *d_body_com,
                                                         const Scalar4 *d_oldvel,
                                                         const Scalar4 *d_vel,
                                                         const unsigned int *d_body,
                                                         BoxDim box,
                                                         Scalar deltaT,
                                                         unsigned int N)
    {
    unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pidx >= N)
        return;

    if (d_body[pidx] != NO_BODY)
        {
        // calculate the virial from the position and velocity from the previous step
        Scalar4 old_vel = d_oldvel[pidx];
        Scalar4 old_pos = d_oldpos[pidx];
        Scalar3 o_p = make_scalar3(old_pos.x, old_pos.y, old_pos.z);
        Scalar4 vel = d_vel[pidx];
        Scalar mass = vel.w;
        Scalar4 net_force = d_net_force[pidx];
        unsigned int bodyIdx = d_body[pidx];
        Scalar3 bodyCOM = make_scalar3(d_body_com[bodyIdx].x,d_body_com[bodyIdx].y,d_body_com[bodyIdx].z);

        Scalar3 rawdr = make_scalar3(o_p.x-bodyCOM.x, o_p.y-bodyCOM.y, o_p.z-bodyCOM.z);
        Scalar3 dr = box.minImage(rawdr);

        Scalar3 fc;
        fc.x = mass * (vel.x - old_vel.x) / deltaT - net_force.x;
        fc.y = mass * (vel.y - old_vel.y) / deltaT - net_force.y;
        fc.z = mass * (vel.z - old_vel.z) / deltaT - net_force.z;

        d_net_virial[0*virial_pitch+pidx] += dr.x * fc.x;
        d_net_virial[1*virial_pitch+pidx] += dr.x * fc.y;
        d_net_virial[2*virial_pitch+pidx] += dr.x * fc.z;
        d_net_virial[3*virial_pitch+pidx] += dr.y * fc.y;
        d_net_virial[4*virial_pitch+pidx] += dr.y * fc.z;
        d_net_virial[5*virial_pitch+pidx] += dr.z * fc.z;
        }
    }

/*! \param d_net_virial Net virial data to update with correction terms
    \param virial_pitch Pitch of d_net_virial
    \param d_net_force Net force on each particle
    \param d_oldpos Old position of particles saved at the start of the step
    \param d_body_com Centres of mass of bodies to which particles belong
    \param d_oldvel Old velocity of particles saved at the start of the step
    \param d_vel Current velocity of particles at the end of the step
    \param d_body Body index of each particle
    \param deltaT Step size
    \param N number of particles in the box
*/
hipError_t gpu_compute_virial_correction_end(Scalar *d_net_virial,
                                              const unsigned int virial_pitch,
                                              const Scalar4 *d_net_force,
                                              const Scalar4 *d_oldpos,
                                              const Scalar4 *d_body_com,
                                              const Scalar4 *d_oldvel,
                                              const Scalar4 *d_vel,
                                              const unsigned int *d_body,
                                              const BoxDim& box,
                                              Scalar deltaT,
                                              unsigned int N)
    {
    assert(d_net_virial);
    assert(d_net_force);
    assert(d_oldpos);
    assert(d_body_com);
    assert(d_oldvel);
    assert(d_vel);

    unsigned int block_size = 192;
    dim3 particle_grid(N/block_size+1, 1, 1);
    dim3 particle_threads(block_size, 1, 1);

    gpu_compute_virial_correction_end_kernel<<<particle_grid, particle_threads>>>(d_net_virial,
                                                                                  virial_pitch,
                                                                                  d_net_force,
                                                                                  d_oldpos,
                                                                                  d_body_com,
                                                                                  d_oldvel,
                                                                                  d_vel,
                                                                                  d_body,
                                                                                  box,
                                                                                  deltaT,
                                                                                  N);

    return hipSuccess;
    }
